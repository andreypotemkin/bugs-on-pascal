
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void check_atomic(float* total)
{
    atomicAdd(total, 1.0f);
}

int main()
{
  float* total;
  hipMallocManaged(&total, sizeof(float));
  *total = 0.0;

  check_atomic<<<1, 32>>>(total);
  hipDeviceSynchronize();

  printf("computed %.1lf, while true is %.1f\n", *total, 32.0);

  hipFree(total);
  return 0;
}
