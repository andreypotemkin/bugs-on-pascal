
#include <hip/hip_runtime.h>
#include <stdio.h>

const int WARP_SIZE = 32;

__global__ void check_shuffle(float* a, float* total)
{
  __shared__ float values[WARP_SIZE];

  values[threadIdx.x] = a[threadIdx.x];
  __syncthreads();

  float val = a[threadIdx.x];

  if (threadIdx.x == 0)
    for (int i = 0; i < WARP_SIZE; i++)
    {
      float v1 =  __shfl(val, i);
      if (v1 != values[i])
        atomicAdd(total, 1);
    }
}


int main()
{
  float* data;
  hipMallocManaged(&data, WARP_SIZE * sizeof(float));
  for (int i = 0; i < WARP_SIZE; i++)
    data[i] = i + 1;

  float* total;
  hipMallocManaged(&total, sizeof(float));
  *total = 0.0;

  check_shuffle<<<1, 32>>>(data, total);
  hipDeviceSynchronize();

  printf("computed %.1lf, while true is %.1f\n", *total, 0.0);

  hipFree(total);
  hipFree(data);
  return 0;
}
